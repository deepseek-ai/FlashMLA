// Copyright (c) 2024, Tri Dao.
// Splitting the different head dimensions to different files to speed up compilation.
// This file is auto-generated. See "generate_kernels.py"

#include "flash_parameter.h"
#include "flash_run_fwd_template_impl.h"
#include <mctlass/numeric_types.h>

template void run_flash_splitkv_fwd_template<
                576,
                32,
                16,
                2,
                true,
                true,
                cutlass::bfloat16_t,
                true,
                512
            >(Flash_fwd_params &params, mcFlashAttn::Flash_launch_params& launch_params,hipStream_t stream);

